#include "hip/hip_runtime.h"
#include "tma_kernels_cuda.h"
#include <cudaTypedefs.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <cuda/barrier>
#include <cuda/ptx>
#include <assert.h>
#include <ATen/cuda/HIPContext.h>

using barrier = cuda::barrier<cuda::thread_scope_block>;
namespace cde = cuda::device::experimental;

#define CUDA_CHECK(call) \
    do { \
        hipError_t status = call; \
        if (status != hipSuccess) { \
            const char* err; \
            hipDrvGetErrorName(status, &err); \
            fprintf(stderr, "CUDA error at line %d in file %s: %s\n", __LINE__, __FILE__, err); \
            exit(1); \
        } \
    } while (0)

#define cdiv(a, b) (((a) + ((b) - 1)) / (b))

__device__ __forceinline__
void tma_load(
    void* __dest, const void* __tensor_map , int __c0, int __c1, ::cuda::barrier<::cuda::thread_scope_block> &__bar)
{
    asm volatile(
        "cp.async.bulk.tensor.2d.shared::cluster.global.tile.mbarrier::complete_tx::bytes "
        "[%0], [%1, {%2, %3}], [%4];\n"
        :
        : "r"(static_cast<_CUDA_VSTD::uint32_t>(__cvta_generic_to_shared(__dest))),
          "l"(__tensor_map),
          "r"(__c0),
          "r"(__c1),
          "r"(static_cast<_CUDA_VSTD::uint32_t>(__cvta_generic_to_shared(::cuda::device::barrier_native_handle(__bar))))
        : "memory");
}

__device__ __forceinline__
void tma_store(
    const void* __tensor_map, int __c0, int __c1, const void* __src)
{
    asm volatile(
        "cp.async.bulk.tensor.2d.global.shared::cta.tile.bulk_group "
        "[%0, {%1, %2}], [%3];\n"
        :
        : "l"(__tensor_map),
          "r"(__c0),
          "r"(__c1),
          "r"(static_cast<_CUDA_VSTD::uint32_t>(__cvta_generic_to_shared(__src)))
        : "memory");
}

// copied from CUTLASS: include/cute/arch/copy_sm90_desc.hpp
__device__ __forceinline__
void tma_descriptor_fence_acquire(const void* desc_ptr)
{
  uint64_t gmem_int_desc = reinterpret_cast<uint64_t>(desc_ptr);
  asm volatile (
    "fence.proxy.tensormap::generic.acquire.gpu [%0], 128;"
    :
    : "l"(gmem_int_desc)
    : "memory");
  asm volatile (
    "cvta.global.u64 %0, %0;"
    :
    : "l"(gmem_int_desc), "l"(gmem_int_desc)
    : "memory");
}

// copied from CUTLASS: include/cute/arch/copy_sm90_desc.hpp
__device__ __forceinline__
void tma_descriptor_fence_release()
{
  asm volatile ("fence.proxy.tensormap::generic.release.gpu;");
}

// copied from CUTLASS: include/cute/arch/util.hpp
__device__ __forceinline__
uint32_t cast_smem_ptr_to_uint(void* ptr)
{
  return static_cast<uint32_t>(__cvta_generic_to_shared(ptr));
}

// copied from CUTLASS: include/cute/arch/copy_sm90_desc.hpp
__device__ __forceinline__
void tma_descriptor_cp_fence_release(void* gmem_desc_ptr, void* smem_desc)
{
  uint64_t gmem_int_desc = reinterpret_cast<uint64_t>(gmem_desc_ptr);
  uint32_t smem_int_desc = cast_smem_ptr_to_uint(smem_desc);
  asm volatile (
    "tensormap.cp_fenceproxy.global.shared::cta.tensormap::generic.release.gpu.sync.aligned [%0], [%1], 128;"
    :: "l"(gmem_int_desc), "r"(smem_int_desc));
}

// copied from CUTLASS: include/cute/arch/cluster_sm90.hpp
__device__ __forceinline__ uint32_t elect_one_sync()
{
  uint32_t pred = 0;
  uint32_t laneid = 0;
  asm volatile(
    "{\n"
    ".reg .b32 %%rx;\n"
    ".reg .pred %%px;\n"
    "     elect.sync %%rx|%%px, %2;\n"
    "@%%px mov.s32 %1, 1;\n"
    "     mov.s32 %0, %%rx;\n"
    "}\n"
    : "+r"(laneid), "+r"(pred)
    : "r"(0xFFFFFFFF));
  return pred;
}

// copied from CUTLASS: include/cutlass/cutlass.h
__device__ __forceinline__ int canonical_warp_idx_sync() { 
    return __shfl_sync(0xffffffff, threadIdx.x / 32, 0);
}

__device__ __forceinline__ void tma_add1_body(
  const void* desc, size_t M, size_t N
) {
    __shared__ alignas(128) float tma_buf[BLOCK_M][BLOCK_N];

  // Calculate coordinates for load / store
  const size_t grid_n = cdiv(N, BLOCK_N);
  const size_t pid_m = blockIdx.x / grid_n;
  const size_t pid_n = blockIdx.x % grid_n;
  const size_t offs_m = pid_m * BLOCK_M;
  const size_t offs_n = pid_n * BLOCK_N;

  // Initialize shared memory barrier with the number of threads participating in the barrier.
  #pragma nv_diag_suppress static_var_with_dynamic_init
  __shared__ barrier bar;

  if (threadIdx.x == 0) {
    // Initialize barrier. All `blockDim.x` threads in block participate.
    init(&bar, blockDim.x);
    // Make initialized barrier visible in async proxy.
    cde::fence_proxy_async_shared_cta();    
  }
  // Syncthreads so initialized barrier is visible to all threads.
  __syncthreads();

  barrier::arrival_token token;
  if (threadIdx.x == 0) {
    // Initiate bulk tensor copy.
    tma_load(tma_buf, desc, offs_n, offs_m, bar);
    // Arrive on the barrier and tell how many bytes are expected to come in.
    token = cuda::device::barrier_arrive_tx(bar, 1, BLOCK_M * BLOCK_N * sizeof(float));
  } else {
    // Other threads just arrive.
    token = bar.arrive();
  }
  // Wait for the data to have arrived.
  bar.wait(std::move(token));

  // Increment all values in the tensor
  static_assert(BLOCK_M == 64);
  static_assert(BLOCK_N == 64);
  const size_t laneIdx = threadIdx.x * 2;
  for (int row = 0; row < BLOCK_M; row++) {
    tma_buf[row][laneIdx] += 1.0;
    tma_buf[row][laneIdx + 1] += 1.0;
  }
  
  // Wait for shared memory writes to be visible to TMA engine.
  cde::fence_proxy_async_shared_cta();
  __syncthreads();
  // After syncthreads, writes by all threads are visible to TMA engine.

  // Initiate TMA transfer to copy shared memory to global memory
  if (threadIdx.x == 0) {
    tma_store(desc, offs_n, offs_m, (void*)tma_buf);
    // Wait for TMA transfer to have finished reading shared memory.
    // Create a "bulk async-group" out of the previous bulk copy operation.
    cde::cp_async_bulk_commit_group();
    // Wait for the group to have completed reading from shared memory.
    cde::cp_async_bulk_wait_group_read<0>();
  }

  // Destroy barrier. This invalidates the memory region of the barrier. If
  // further computations were to take place in the kernel, this allows the
  // memory location of the shared memory barrier to be reused.
  if (threadIdx.x == 0) {
    (&bar)->~barrier();
  }
}

TmaDesc::TmaDesc(float* gmem_ptr, size_t M, size_t N) {
  // Note: fastest-moving dimension always comes first here
  constexpr uint32_t rank = 2;
  const uint64_t size[rank] = {N, M}; // elements
  const uint64_t stride[rank - 1] = {N * sizeof(float)}; // bytes
  const uint32_t box_size[rank] = {BLOCK_N, BLOCK_M}; // elements
  const uint32_t elem_stride[rank] = {1, 1}; // elements

  CUDA_CHECK(cuTensorMapEncodeTiled(
      &desc_,
      CUtensorMapDataType::CU_TENSOR_MAP_DATA_TYPE_FLOAT32,
      rank,
      (void*)gmem_ptr,
      size,
      stride,
      box_size,
      elem_stride,
      CUtensorMapInterleave::CU_TENSOR_MAP_INTERLEAVE_NONE,
      CUtensorMapSwizzle::CU_TENSOR_MAP_SWIZZLE_NONE,
      CUtensorMapL2promotion::CU_TENSOR_MAP_L2_PROMOTION_L2_128B,
      CUtensorMapFloatOOBfill::CU_TENSOR_MAP_FLOAT_OOB_FILL_NONE
  ));
}

// This forces exactly 1 warp per SM, which is useful for benchmarking latency
template<typename T>
size_t maximize_smem_usage(T kernel) {
    hipFuncAttributes attr;
    hipFuncGetAttributes(&attr, reinterpret_cast<const void*>(kernel));
    const size_t dynamicSharedSize = 227 * 1024 - attr.sharedSizeBytes;
    hipFuncSetAttribute(reinterpret_cast<const void*>(kernel), hipFuncAttributeMaxDynamicSharedMemorySize, dynamicSharedSize);
    return dynamicSharedSize;
}

__global__ void grid_constant_kernel(
    const __grid_constant__ CUtensorMap desc,
    size_t M, size_t N)
{
  tma_add1_body(&desc, M, N);
}

void launch_grid_constant_kernel(float* tensor, size_t M, size_t N)
{
    TmaDesc desc(tensor, M, N);
    dim3 threadsPerBlock(32, 1, 1);
    dim3 numBlocks(cdiv(M, BLOCK_M) * cdiv(N, BLOCK_N), 1, 1);
    const size_t dynamicSharedSize = maximize_smem_usage(grid_constant_kernel);
    const hipStream_t stream = at::cuda::getCurrentCUDAStream();
    grid_constant_kernel<<<numBlocks, threadsPerBlock, dynamicSharedSize, stream>>>(*desc.get(), M, N);
}

__global__ void fence_kernel(
    uint8_t* desc_gmem_ptr,
    size_t M, size_t N)
{
  tma_descriptor_fence_acquire(desc_gmem_ptr);
  tma_add1_body((void*)desc_gmem_ptr, M, N);
}

void launch_fence_kernel(uint8_t* desc, size_t M, size_t N)
{
    dim3 threadsPerBlock(32, 1, 1);
    dim3 numBlocks(cdiv(M, BLOCK_M) * cdiv(N, BLOCK_N), 1, 1);
    const size_t dynamicSharedSize = maximize_smem_usage(fence_kernel);
    fence_kernel<<<numBlocks, threadsPerBlock, dynamicSharedSize>>>(desc, M, N);
}

__global__ void ondevice_kernel(
    uint8_t* desc_gmem_ptr,
    size_t M, size_t N)
{
  if (canonical_warp_idx_sync() == 0 && elect_one_sync()) {
    // Here is where we would set up the descriptor with a GMEM write
    // It turns out that even without any GMEM mutation, the release fence takes at least 1000 cycles
    // Before benchmarking, initialize desc_gmem_ptr to point to a valid tensormap
    
    // After the GMEM write, we need a release fence
    tma_descriptor_fence_release();
    
    // Followed by an acquire fence
    tma_descriptor_fence_acquire(desc_gmem_ptr);
  }
  // we don't need __syncthreads() due to __syncthreads() in tma_add1_body()
  tma_add1_body((void*)desc_gmem_ptr, M, N);
}

void launch_ondevice_kernel(uint8_t* desc, size_t M, size_t N)
{
    dim3 threadsPerBlock(32, 1, 1);
    dim3 numBlocks(cdiv(M, BLOCK_M) * cdiv(N, BLOCK_N), 1, 1);
    const size_t dynamicSharedSize = maximize_smem_usage(ondevice_kernel);
    const hipStream_t stream = at::cuda::getCurrentCUDAStream();
    ondevice_kernel<<<numBlocks, threadsPerBlock, dynamicSharedSize, stream>>>(desc, M, N);
}

__device__ alignas(128) uint8_t arr_g[128];

__global__ void ondevice_cpfence_kernel(
    const __grid_constant__ CUtensorMap desc,
    uint8_t* desc_gmem_ptr,
    size_t M, size_t N)
{
    __shared__ alignas(128) uint32_t desc_smem_ptr[32];
    if (threadIdx.x < 32) {
      // Attempt to copy the desc I passed in through grid const into shared mem
      void* desc_param_ptr = (void*)(&desc);
      desc_smem_ptr[threadIdx.x] =((uint32_t*)desc_param_ptr)[threadIdx.x];
      __syncwarp();
      
      // Attempt fused copy + release fence into GMEM. Currently this raises an error:
      //   CUDA error: an illegal instruction was encountered
      // To repro:
      //    `python setup.py develop`
      //    `python test.py`
      //
      // Notes
      // * This error doesn't happen during build, it happens at runtime. Also, I made sure that my build is using CUDA 12.4 nvcc
      //
      // * I am on an old driver (525), but I was able to run the CUTLASS 57_hopper_grouped_gemm example with no problems.
      //   The code here is copied from the CUTLASS Hopper grouped gemm. So I don't think it is a driver problem.
      //
      // * If I comment out this line, `python test.py` passes. Note that desc_gmem_ptr is initialized to point at a valid TMA
      //   tensormap in the test code (the test code copies over the same data to desc_gmem_ptr as it passes through __grid_constant__)
      tma_descriptor_cp_fence_release(desc_gmem_ptr, desc_smem_ptr);

      tma_descriptor_fence_acquire(desc_gmem_ptr);
    }
    // we don't need __syncthreads() due to __syncthreads() in tma_add1_body()
    tma_add1_body(desc_gmem_ptr, M, N);
}

void launch_ondevice_cpfence_kernel(float* tensor, uint8_t* desc_gmem_ptr, size_t M, size_t N)
{
    TmaDesc desc(tensor, M, N);
    dim3 threadsPerBlock(32, 1, 1);
    dim3 numBlocks(cdiv(M, BLOCK_M) * cdiv(N, BLOCK_N), 1, 1);
    const size_t dynamicSharedSize = maximize_smem_usage(ondevice_cpfence_kernel);
    const hipStream_t stream = at::cuda::getCurrentCUDAStream();
    ondevice_cpfence_kernel<<<numBlocks, threadsPerBlock, dynamicSharedSize, stream>>>(*desc.get(), desc_gmem_ptr, M, N);
}
